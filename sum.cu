
#include <hip/hip_runtime.h>
#include<iostream>

const int n = 4096;
const int m = 2048;

//kernel
__global__ void kernelmatsum(float* A , float*B,float*C, int m,int n)
{
    // int i = blockDim.x*blockIdx.x+threadIdx.x;
    // int j = blockDim.y*blockIdx.y+threadIdx.y;

    // C[blockDim.x*j+i]=A[blockDim.x*j+i]+B[blockDim.x*j+i];

    int r = blockDim.y * blockIdx.y + threadIdx.y;
    int c = blockDim.x * blockIdx.x + threadIdx.x;

    C[r * n + c] = A[r * n + c] + B[r * n + c];
    
}

void metadd(float* A_h,float*B_h,float*C_h,int m,int n){

    float* A_d,*B_d,*C_d;
    int size = m*n*sizeof(float);

    hipMalloc((void**)&A_d,size);
    hipMalloc((void**)&B_d,size);
    hipMalloc((void**)&C_d,size);

    hipMemcpy(A_d,A_h,size,hipMemcpyHostToDevice);
    hipMemcpy(B_d,B_h,size,hipMemcpyHostToDevice);
    uint threadsX = 32, threadsY = 16, blocksX = ceil(n / threadsX), blocksY = ceil(m / threadsY);
    dim3 blocks(blocksX, blocksY), threads(threadsX, threadsY);
    kernelmatsum<<<blocks,threads>>>(A_d,B_d,C_d,m,n);

    hipMemcpy(C_h,C_d,size,hipMemcpyDeviceToHost);

    hipFree(&A_d);
    hipFree(&B_d);
    hipFree(&C_d);



}

int main(){



float * A_h = (float*)malloc(m*n*sizeof(float));
float * B_h = (float*)malloc(m*n*sizeof(float));
float * C_h = (float*)malloc(m*n*sizeof(float));

int sum = 0;
for (int  i = 0; i < m*n; i++)
{
    A_h[i]=sum;
    B_h[i]=sum;
    sum++;
    // std::cout << 2 * sum << std::endl;

}
metadd(A_h,B_h,C_h,m,n);
sum = 0;
for (int  i = 0; i < m*n; i++)
{
    /* code */
    // printf(" %0.0f  ",C_h[i]);
    if(i * 2 != C_h[i]) std::cout << " Error at " << i << std::endl;
    //std::cout<<C_h[i]<<"  ";
}







    

    
    return 0;
}